#include "hip/hip_runtime.h"
// Fast Accurate Fourier Transform (FAFT) was written by Oscar R. Cabrera L.
// Contributors: Renan Cabrera, Denys I. Bondar.

#include "FAFTp_R2C_C2R.h"

// axSplit 0

__global__ void IFAFT128_C2R_ax0_dev( float *re, float *im, float2 *data65, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
    
    re += ((sector>>6)<<12) + (sector & 63) + (tid<<6);
    im += ((sector>>6)<<12) + (sector & 63) + (tid<<6) + (1<<11);
    if (tid == 0) data65 += sector;

    float2 y[16];
    
    load128_half_C2R_ax0( 8, y, re, im, data65, 16, tid );
    
    GENERAL_FAFT128( y, dx, delta, segment, tid );
        
    store128_half_C2R_ax0<8>( y, re, data65, 16, tid );
}

__global__ void IFAFT128_C2C_ax1_axSplit0_dev( float *re, float *im, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
    
    re += ((sector>>5)<<11) + (sector<<6) + tid;
    im += ((sector>>5)<<11) + (sector<<6) + tid  + (1<<11);
	
    float2 y[16];
    
    load128_half_C2C_ax1_axSplit0( 8, y, re, im, 16 );
    
    GENERAL_FAFT128( y, dx, delta, segment, tid );
        
    store128_half_C2C_ax1_axSplit0<8>( y, re, im, 16 );
}

__global__ void IFAFT128_C2C_65_ax1_axSplit0_dev( float2 *data65, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
    
    data65 += (sector<<6) + tid;	

    float2 y[16];
    
    load128_half_C2C_65_ax1_axSplit0( 8, y, data65, 16 );
    
    GENERAL_FAFT128( y, dx, delta, segment, tid );
        
    store128_half_C2C_65_ax1_axSplit0<8>( y, data65, 16 );
}

__global__ void IFAFT128_C2C_ax2_axSplit0_dev( float *re, float *im, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
    
    re += ((sector>>11)<<18) + (sector & ((1<<11) - 1)) + (tid<<12);
    im += ((sector>>11)<<18) + (sector & ((1<<11) - 1)) + (tid<<12) + (1<<11);
	
    float2 y[16];
    
    load128_half_C2C_ax2_axSplit0( 8, y, re, im, 16 );
    
    GENERAL_FAFT128( y, dx, delta, segment, tid );
			
    store128_half_C2C_ax2_axSplit0<8>( y, re, im, 16 );
}

__global__ void IFAFT128_C2C_65_ax2_axSplit0_dev( float2 *data65, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
        
    data65 += ((sector>>6)<<12) + (sector & 63) + (tid<<6);		
	
    float2 y[16];
    
    load128_half_C2C_65_ax2_axSplit0( 8, y, data65, 16 );
			
    GENERAL_FAFT128( y, dx, delta, segment, tid );

    store128_half_C2C_65_ax2_axSplit0<8>( y, data65, 16 );
}

// >>>

__global__ void IFAFT128_C2C_ax3_axSplit0_dev( float *re, float *im, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
           
    re += ((sector>>11)<<11) + sector + (tid<<18);
    im += ((sector>>11)<<11) + sector + (tid<<18) + (1<<11);
	
    float2 y[16];
    
    load128_half_C2C_ax3_axSplit0( 8, y, re, im, 16 );
			
    GENERAL_FAFT128( y, dx, delta, segment, tid );

    store128_half_C2C_ax3_axSplit0<8>( y, re, im, 16 );
}

__global__ void IFAFT128_C2C_65_ax3_axSplit0_dev( float2 *data65, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
    
    data65 += ((sector>>12)<<18) + (sector & ((1<<12)-1)) + (tid<<12);				
	
    float2 y[16];
    
    load128_half_C2C_65_ax3_axSplit0( 8, y, data65, 16 );
			
    GENERAL_FAFT128( y, dx, delta, segment, tid );

    store128_half_C2C_65_ax3_axSplit0<8>( y, data65, 16 );
}

// ***************************
// ax Split 1

__global__ void IFAFT128_C2R_ax1_dev( float *re, float *im, float2 *data65, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;

    re += (sector*64) + tid;
    im += (sector*64) + tid + 32;
    if (tid == 0) data65 += sector;

    float2 y[16];
    
    load128_half_C2R_ax1( 8, y, re, im, data65, 16, tid );
    
    GENERAL_FAFT128( y, dx, delta, segment, tid );
    
    store128_half_C2R_ax1<8>( y, re, data65, 16, tid );
}

__global__ void IFAFT128_C2C_ax0_axSplit1_dev( float *re, float *im, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
    
    re += ((sector>>5)<<12) + (sector & ((1<<5)-1)) + (tid<<6);
    im += ((sector>>5)<<12) + (sector & ((1<<5)-1)) + (tid<<6) + 32;
	
    float2 y[16];
    
    load128_half_C2C_ax0_axSplit1( 8, y, re, im, 16 );
    
    GENERAL_FAFT128( y, dx, delta, segment, tid );
        
    store128_half_C2C_ax0_axSplit1<8>( y, re, im, 16 );
}

__global__ void IFAFT128_C2C_65_ax0_axSplit1_dev( float2 *data65, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
        
    data65 += (sector<<6) + tid;		
	
    float2 y[16];
    
    load128_half_C2C_65_ax0_axSplit1( 8, y, data65, 16 );
    
    GENERAL_FAFT128( y, dx, delta, segment, tid );
        
    store128_half_C2C_65_ax0_axSplit1<8>( y, data65, 16 );
}

__global__ void IFAFT128_C2C_ax2_axSplit1_dev( float *re, float *im, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
    
    re += ((sector>>11)<<18) - ((sector>>11)<<11) + ((sector>>5)<<5) + (sector & ((1<<11) - 1)) + (tid<<12);		
    im += ((sector>>11)<<18) - ((sector>>11)<<11) + ((sector>>5)<<5) + (sector & ((1<<11) - 1)) + (tid<<12) + 32;
    		
    float2 y[16];
    
    load128_half_C2C_ax2_axSplit1( 8, y, re, im, 16 );
			
    GENERAL_FAFT128( y, dx, delta, segment, tid );

    store128_half_C2C_ax2_axSplit1<8>( y, re, im, 16 );
}

__global__ void IFAFT128_C2C_65_ax2_axSplit1_dev( float2 *data65, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
        
    data65 += ((sector>>6)<<12) + (sector & 63) + (tid<<6);		
	
    float2 y[16];
    
    load128_half_C2C_65_ax2_axSplit1( 8, y, data65, 16 );
			
    GENERAL_FAFT128( y, dx, delta, segment, tid );

    store128_half_C2C_65_ax2_axSplit1<8>( y, data65, 16 );
}

// >>>

__global__ void IFAFT128_C2C_ax3_axSplit1_dev( float *re, float *im, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
	    
    re += ((sector>>5)<<5) + sector + (tid<<18);   		
    im += ((sector>>5)<<5) + sector + (tid<<18) + 32;
    		
    float2 y[16];
    
    load128_half_C2C_ax3_axSplit1( 8, y, re, im, 16 );
			
    GENERAL_FAFT128( y, dx, delta, segment, tid );
    
    store128_half_C2C_ax3_axSplit1<8>( y, re, im, 16 );
}

__global__ void IFAFT128_C2C_65_ax3_axSplit1_dev( float2 *data65, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
    
    data65 += ((sector>>12)<<18) + (sector & ((1<<12)-1)) + (tid<<12);		
	
    float2 y[16];
    
    load128_half_C2C_65_ax3_axSplit1( 8, y, data65, 16 );
			
    GENERAL_FAFT128( y, dx, delta, segment, tid );
    
    store128_half_C2C_65_ax3_axSplit1<8>( y, data65, 16 );
}

// <<<

// ***************************
// ax Split 2

__global__ void IFAFT128_C2R_ax2_dev( float *re, float *im, float2 *data65, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
   
    re += ((sector>>12)<<18) + (sector & ((1<<12)-1)) + (tid<<12);
    im += ((sector>>12)<<18) + (sector & ((1<<12)-1)) + (tid<<12) + (1<<17);
    if (tid == 0) data65 += sector;
	
    float2 y[16];
    
    load128_half_C2R_ax2( 8, y, re, im, data65, 16, tid );
			
    GENERAL_FAFT128( y, dx, delta, segment, tid );
    
    store128_half_C2R_ax2<8>( y, re, data65, 16, tid );
}

__global__ void IFAFT128_C2C_ax0_axSplit2_dev( float *re, float *im, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
        		
    re += ((sector>>11)<<17) + ((sector>>6)<<12) + (sector & ((1<<6)-1)) + (tid<<6);    		
    im += ((sector>>11)<<17) + ((sector>>6)<<12) + (sector & ((1<<6)-1)) + (tid<<6) + (1<<17);

    float2 y[16];
    
    load128_half_C2C_ax0_axSplit2( 8, y, re, im, 16 );

    GENERAL_FAFT128( y, dx, delta, segment, tid );		

    store128_half_C2C_ax0_axSplit2<8>( y, re, im, 16 );
}

__global__ void IFAFT128_C2C_65_ax0_axSplit2_dev( float2 *data65, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
        	
    data65 += ((sector>>6)<<12) + (sector & 63) + (tid<<6);		
	
    float2 y[16];    

    load128_half_C2C_65_ax0_axSplit2( 8, y, data65, 16 );
    
    GENERAL_FAFT128( y, dx, delta, segment, tid );
    		
    store128_half_C2C_65_ax0_axSplit2<8>( y, data65, 16 );    
}

__global__ void IFAFT128_C2C_ax1_axSplit2_dev( float *re, float *im, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
	    
    re += ((sector>>11)<<17) + (sector<<6) + tid;
    im += ((sector>>11)<<17) + (sector<<6) + tid + (1<<17);
    		
    float2 y[16];
    
    load128_half_C2C_ax1_axSplit2( 8, y, re, im, 16 );
    
    GENERAL_FAFT128( y, dx, delta, segment, tid );
    
    store128_half_C2C_ax1_axSplit2<8>( y, re, im, 16 );
}

__global__ void IFAFT128_C2C_65_ax1_axSplit2_dev( float2 *data65, float dx, float delta, int segment )
{
    int tid = threadIdx.x;    
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
        
    data65 += (sector<<6) + tid;		
	
    float2 y[16];
    
    load128_half_C2C_65_ax1_axSplit2( 8, y, data65, 16 );
			
    GENERAL_FAFT128( y, dx, delta, segment, tid );
    
    store128_half_C2C_65_ax1_axSplit2<8>( y, data65, 16 );
}

// >>>

__global__ void IFAFT128_C2C_ax3_axSplit2_dev( float *re, float *im, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
    
    re += sector + (tid<<18);
    im += sector + (tid<<18) + (1<<17);
    		
    float2 y[16];
    
    load128_half_C2C_ax3_axSplit2( 8, y, re, im, 16 );
			
    GENERAL_FAFT128( y, dx, delta, segment, tid );

    store128_half_C2C_ax3_axSplit2<8>( y, re, im, 16 );
}

__global__ void IFAFT128_C2C_65_ax3_axSplit2_dev( float2 *data65, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
    
    data65 += ((sector>>12)<<18) + (sector & ((1<<12)-1)) + (tid<<12);
	
    float2 y[16];
    
    load128_half_C2C_65_ax3_axSplit2( 8, y, data65, 16 );
			
    GENERAL_FAFT128( y, dx, delta, segment, tid );
    
    store128_half_C2C_65_ax3_axSplit2<8>( y, data65, 16 );
}

// <<<

// ***************************
// ax Split 3

__global__ void IFAFT128_C2R_ax3_dev( float *re, float *im, float2 *data65, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
   
    re += sector + (tid<<18);
    im += sector + (tid<<18) + (1<<23);
    if (tid == 0) data65 += sector;      
    
    float2 y[16];
    
    load128_half_C2R_ax3( 8, y, re, im, data65, 16, tid );
    
    GENERAL_FAFT128( y, dx, delta, segment, tid );

    store128_half_C2R_ax3<8>( y, re, data65, 16, tid );
}

__global__ void IFAFT128_C2C_ax0_axSplit3_dev( float *re, float *im, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
    
    re += ((sector>>6)<<12) + (sector & 63) + (tid<<6);
    im += ((sector>>6)<<12) + (sector & 63) + (tid<<6) + (1<<23);

    float2 y[16];
    
    load128_half_C2C_ax0_axSplit3( 8, y, re, im, 16 );
    
    GENERAL_FAFT128( y, dx, delta, segment, tid );
    
    store128_half_C2C_ax0_axSplit3<8>( y, re, im, 16 );    
}

__global__ void IFAFT128_C2C_65_ax0_axSplit3_dev( float2 *data65, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
        	
    data65 += ((sector>>6)<<12) + (sector & 63) + (tid<<6);	
	
    float2 y[16];

    load128_half_C2C_65_ax0_axSplit3( 8, y, data65, 16 );
    
    GENERAL_FAFT128( y, dx, delta, segment, tid );
			
    store128_half_C2C_65_ax0_axSplit3<8>( y, data65, 16 );
}

__global__ void IFAFT128_C2C_ax1_axSplit3_dev( float *re, float *im, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
    
    re += (sector<<6) + tid;
    im += (sector<<6) + tid + (1<<23);  
    		
    float2 y[16];
    
    load128_half_C2C_ax1_axSplit3( 8, y, re, im, 16 );
			
    GENERAL_FAFT128( y, dx, delta, segment, tid );
    
    store128_half_C2C_ax1_axSplit3<8>( y, re, im, 16 );
}

__global__ void IFAFT128_C2C_65_ax1_axSplit3_dev( float2 *data65, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
            
    data65 += (sector<<6) + tid;
	
    float2 y[16];
    
    load128_half_C2C_65_ax1_axSplit3( 8, y, data65, 16 );
			
    GENERAL_FAFT128( y, dx, delta, segment, tid );

    store128_half_C2C_65_ax1_axSplit3<8>( y, data65, 16 );
}

__global__ void IFAFT128_C2C_ax2_axSplit3_dev( float *re, float *im, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
    
    re += ((sector>>12)<<18) + (sector & ((1<<12)-1)) + (tid<<12);
    im += ((sector>>12)<<18) + (sector & ((1<<12)-1)) + (tid<<12) + (1<<23);
    		
    float2 y[16];
    
    load128_half_C2C_ax2_axSplit3( 8, y, re, im, 16 );
			
    GENERAL_FAFT128( y, dx, delta, segment, tid );

    store128_half_C2C_ax2_axSplit3<8>( y, re, im, 16 );
}

__global__ void IFAFT128_C2C_65_ax2_axSplit3_dev( float2 *data65, float dx, float delta, int segment )
{
    int tid = threadIdx.x;
    size_t sector = blockIdx.y*gridDim.x + blockIdx.x;
        
    data65 += ((sector>>12)<<18) + (sector & ((1<<12)-1)) + (tid<<12);
	
    float2 y[16];
    
    load128_half_C2C_65_ax2_axSplit3( 8, y, data65, 16 );
			
    GENERAL_FAFT128( y, dx, delta, segment, tid );

    store128_half_C2C_65_ax2_axSplit3<8>( y, data65, 16 );
}


extern "C" int IFAFT128_4D_C2R( float *data, float2 *data65, float beta, float delta, int segment, int axes, int makeC2R, int axesSplit )
{
	int success = 1;
	
	dim3 grid_C2R(64*64, 64);
	dim3 grid_C2C(64*64/2, 64);
	dim3 grid_C2C_65(64*64, 1);
	
	switch(axes)
	{
		case 0:
			if (makeC2R == 1){			
				IFAFT128_C2R_ax0_dev<<< grid_C2R, 16 >>>( data, data, data65, beta, delta, segment );
			}
			else{
				switch(axesSplit)
				{
					case 0:
						success = 0;
						break;
						
					case 1:
						IFAFT128_C2C_ax0_axSplit1_dev<<< grid_C2C, 16 >>>( data, data, beta, delta, segment );
				                IFAFT128_C2C_65_ax0_axSplit1_dev<<< grid_C2C_65, 16 >>>( data65, beta, delta, segment );					
						break;
						
					case 2:
						IFAFT128_C2C_ax0_axSplit2_dev<<< grid_C2C, 16 >>>( data, data, beta, delta, segment );
				                IFAFT128_C2C_65_ax0_axSplit2_dev<<< grid_C2C_65, 16 >>>( data65, beta, delta, segment );					
						break;
						
					case 3:
						IFAFT128_C2C_ax0_axSplit3_dev<<< grid_C2C, 16 >>>( data, data, beta, delta, segment );
				                IFAFT128_C2C_65_ax0_axSplit3_dev<<< grid_C2C_65, 16 >>>( data65, beta, delta, segment );					
						break;
					
					default:
						success = 0;
						break;				
				}
			}
			
			break;		
			
		case 1:
			if (makeC2R == 1){
				IFAFT128_C2R_ax1_dev<<< grid_C2R, 16 >>>( data, data, data65, beta, delta, segment  );
			}
			else{
				switch(axesSplit)
				{
					case 0:
						IFAFT128_C2C_ax1_axSplit0_dev<<< grid_C2C, 16 >>>( data, data, beta, delta, segment );
						IFAFT128_C2C_65_ax1_axSplit0_dev<<< grid_C2C_65, 16 >>>( data65, beta, delta, segment );					
						break;
					
					case 1:
						success = 0;
						break;
						
					case 2:
						IFAFT128_C2C_ax1_axSplit2_dev<<< grid_C2C, 16 >>>( data, data, beta, delta, segment );
						IFAFT128_C2C_65_ax1_axSplit2_dev<<< grid_C2C_65, 16 >>>( data65, beta, delta, segment );					
						break;
						
					case 3:
						IFAFT128_C2C_ax1_axSplit3_dev<<< grid_C2C, 16 >>>( data, data, beta, delta, segment );
						IFAFT128_C2C_65_ax1_axSplit3_dev<<< grid_C2C_65, 16 >>>( data65, beta, delta, segment );					
						break;
				
					default:
						success = 0;
						break;		
				}
			}
			
			break;

		case 2:
			if (makeC2R == 1){
				IFAFT128_C2R_ax2_dev<<< grid_C2R, 16 >>>( data, data, data65, beta, delta, segment );
			}
			else{
				switch(axesSplit)
				{
					case 0:
						IFAFT128_C2C_ax2_axSplit0_dev<<< grid_C2C, 16 >>>( data, data, beta, delta, segment );
						IFAFT128_C2C_65_ax2_axSplit0_dev<<< grid_C2C_65, 16 >>>( data65, beta, delta, segment );					
						break;
						
					case 1:
						IFAFT128_C2C_ax2_axSplit1_dev<<< grid_C2C, 16 >>>( data, data, beta, delta, segment );
						IFAFT128_C2C_65_ax2_axSplit1_dev<<< grid_C2C_65, 16 >>>( data65, beta, delta, segment );					
						break;
						
					case 2:
						success = 0;
						break;
						
					case 3:
						IFAFT128_C2C_ax2_axSplit3_dev<<< grid_C2C, 16 >>>( data, data, beta, delta, segment );
						IFAFT128_C2C_65_ax2_axSplit3_dev<<< grid_C2C_65, 16 >>>( data65, beta, delta, segment );					
						break;
				
					default:
						success = 0;
						break;		
				}
			}
			
			break;
			
		case 3:
			if (makeC2R == 1){
				IFAFT128_C2R_ax3_dev<<< grid_C2R, 16 >>>( data, data, data65, beta, delta, segment );
			}
			else{
				switch(axesSplit)
				{
					case 0:
						IFAFT128_C2C_ax3_axSplit0_dev<<< grid_C2C, 16 >>>( data, data, beta, delta, segment );
						IFAFT128_C2C_65_ax3_axSplit0_dev<<< grid_C2C_65, 16 >>>( data65, beta, delta, segment );					
						break;
						
					case 1:
						IFAFT128_C2C_ax3_axSplit1_dev<<< grid_C2C, 16 >>>( data, data, beta, delta, segment );
						IFAFT128_C2C_65_ax3_axSplit1_dev<<< grid_C2C_65, 16 >>>( data65, beta, delta, segment );					
						break;
						
					case 2:
						IFAFT128_C2C_ax3_axSplit2_dev<<< grid_C2C, 16 >>>( data, data, beta, delta, segment );
						IFAFT128_C2C_65_ax3_axSplit2_dev<<< grid_C2C_65, 16 >>>( data65, beta, delta, segment );
						break;
						
					case 3:
						success = 0;
						break;
				
					default:
						success = 0;
						break;		
				}
			}
			
			break;

		default:
			success = 0;
			break;
	}
		
	hipDeviceSynchronize();
	
	return success;
}

